#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include "real.h"

#include "parallelSpmv.h"

#define FATAL(msg) \
    do {\
        fprintf(stderr, "[%s:%d] %s\n", __FILE__, __LINE__, msg);\
        exit(-1);\
    } while(0)

#define MAXTHREADS 128
#define REP 1000

#ifdef DOUBLE
    texture<int2>  xTex;
    texture<int2>  valTex;
#else
    texture<float> xTex;
    texture<float> valTex;
#endif

void meanAndSd(real *mean, real *sd,real *data, int n)
{
    real sum = (real) 0.0; 
    real standardDeviation = (real) 0.0;

    for(int i=0; i<n; ++i) {
        sum += data[i];
    } // end for //

    *mean = sum/n;

    for(int i=0; i<n; ++i) {
        standardDeviation += pow(data[i] - *mean, 2);
    } // end for //
    *sd=sqrt(standardDeviation/n);
} // end of calculateSD //


int main(int argc, char *argv[]) 
{
    #include "parallelSpmvData.h"

    // verifing number of input parameters //
   char exists='t';
   char checkSol='f';
    
    if (argc < 3 ) {
        printf("Use: %s  Matrix_filename InputVector_filename  [SolutionVector_filename  [# of streams] ]  \n", argv[0]);     
        exists='f';
    } // endif //
    
    FILE *fh=NULL;
    // testing if matrix file exists
    if((fh = fopen(argv[1], "rb")  )   == NULL) {
        printf("No matrix file found.\n");
        exists='f';
    } // end if //
    
    // testing if input file exists
    if((fh = fopen(argv[2], "rb")  )   == NULL) {
        printf("No input vector file found.\n");
        exists='f';
    } // end if //

    // testing if output file exists
    if (argc  >3 ) {
        if((fh = fopen(argv[3], "rb")  ) == NULL) {
            printf("No output vector file found.\n");
            exists='f';
        } else {
            checkSol='t';
        } // end if //
    } // end if //

    if (argc  > 4 ) {
        nStreams = atoi(argv[4]);
    } // end if //
    
    if (fh) fclose(fh);
    
    if (exists == 'f') {
        printf("Quitting.....\n");
        exit(0);
    } // end if //
    
    printf("Solving using %d streams\n", nStreams);

    stream= (hipStream_t *) malloc(sizeof(hipStream_t) * nStreams);
    
    starRow = (int *) malloc(sizeof(int) * nStreams+1); 
    starRow[0]=0;
    reader(&n_global,&nnz_global, starRow, 
           &row_ptr,&col_idx,&val,
           argv[1], nStreams);
    
    
    // ready to start //    
    hipError_t cuda_ret;
    
    real *w=NULL;
    real *v=NULL; // <-- input vector to be shared later
    //real *v_off=NULL; // <-- input vector to be shared later
    
    
    v     = (real *) malloc(n_global*sizeof(real));
    w     = (real *) malloc(n_global*sizeof(real)); 

    // reading input vector
    vectorReader(v, &n_global, argv[2]);
//////////////////////////////////////
// cuda stuff start here


    int *rows_d, *cols_d;
    real *vals_d;
    real *v_d, *w_d;

    // Allocating device memory for input matrices 

    cuda_ret = hipMalloc((void **) &rows_d,  (n_global+1)*sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for rows_d");
    
    cuda_ret = hipMalloc((void **) &cols_d,  (nnz_global)*sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for cols_d");
    
    cuda_ret = hipMalloc((void **) &vals_d,  (nnz_global)*sizeof(real));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for vals_d");

    cuda_ret = hipMalloc((void **) &v_d,  (n_global)*sizeof(real));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for x_d");

    cuda_ret = hipMalloc((void **) &w_d,  (n_global)*sizeof(real));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for y_d");

   // Copy the input matrices from the host memory to the device memory

    cuda_ret = hipMemcpy(rows_d, row_ptr, (n_global+1)*sizeof(int),hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix rows_d");

    cuda_ret = hipMemcpy(cols_d, col_idx, (nnz_global)*sizeof(int),hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix cols_d");

    cuda_ret = hipMemcpy(vals_d, val, (nnz_global)*sizeof(real),hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix vals_d");


    cuda_ret = hipMemcpy(v_d, v, (n_global)*sizeof(real),hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix x_d");


    const int basicSize = 32;
    const real parameter2Adjust = 0.5;

    meanNnzPerRow = (real*) malloc(nStreams*sizeof(real));
    sd            = (real*) malloc(nStreams*sizeof(real ));
    block = (dim3 *) malloc(nStreams*sizeof(dim3 )); 
    grid  = (dim3 *) malloc(nStreams*sizeof(dim3 )); 
    sharedMemorySize = (size_t *) calloc(nStreams, sizeof(size_t)); 

    for (int s=0; s<nStreams; ++s) {
        block[s].x = basicSize;
        block[s].y = 1;
        block[s].z = 1;
        grid[s].x = 1;
        grid[s].y = 1;
        grid[s].z = 1;
    } // end for //




    for (int s=0; s<nStreams; ++s) {
        int nrows = starRow[s+1]-starRow[s];
        /////////////////////////////////////////////////////
        // determining the standard deviation of the nnz per row
        real *temp=(real *) calloc(nrows,sizeof(real));
        
        for (int row=starRow[s], i=0; row<starRow[s]+nrows; ++row, ++i) {
            temp[i] = row_ptr[row+1] - row_ptr[row];
        } // end for //
        meanAndSd(&meanNnzPerRow[s],&sd[s],temp, nrows);
        //printf("file: %s, line: %d, gpu on-prcoc:   %d, mean: %7.3f, sd: %7.3f using: %s\n", __FILE__, __LINE__, s , meanNnzPerRow[s], sd[s], (meanNnzPerRow[s] + 0.5*sd[s] < 32) ? "spmv0": "spmv1" );
        free(temp);
        /////////////////////////////////////////////////////

        //cuda_ret = hipStreamCreateWithFlags(&stream0[gpu], hipStreamDefault);
        cuda_ret = hipStreamCreateWithFlags(&stream[s], hipStreamNonBlocking ) ;
        if(cuda_ret != hipSuccess) FATAL("Unable to create stream0 ");
        
        printf("In Stream: %d\n",s);
        if (meanNnzPerRow[s] + parameter2Adjust*sd[s] < basicSize) {
        	// these mean use scalar spmv
            grid[s].x = (   (  nrows + block[s].x -1) /block[s].x );
            printf("using scalar spmv for on matrix,  blockSize: [%d, %d] %f, %f\n",block[s].x,block[s].y, meanNnzPerRow[s], sd[s]) ;
        } else {
            // these mean use vector spmv
            if (meanNnzPerRow[s] >= 2*basicSize) {
                block[s].x = 2*basicSize;
            } // end if //
            block[s].y=MAXTHREADS/block[s].x;
            grid[s].x = ( (nrows + block[s].y - 1) / block[s].y ) ;
        	sharedMemorySize[s]=block[s].x*block[s].y*sizeof(real);
            printf("using vector spmv for on matrix,  blockSize: [%d, %d] %f, %f\n",block[s].x,block[s].y, meanNnzPerRow[s], sd[s]) ;
        } // end if // 

    } // end for //

    // Timing should begin here//
    struct timeval tp;                                   // timer
    double elapsed_time;
    
    gettimeofday(&tp,NULL);  // Unix timer
    elapsed_time = -(tp.tv_sec*1.0e6 + tp.tv_usec);
    for (int t=0; t<REP; ++t) {

        cuda_ret = hipMemset(w_d, 0, (size_t) n_global*sizeof(real) );
        if(cuda_ret != hipSuccess) FATAL("Unable to set device for matrix w_d");
        
        
        for (int s=0; s<nStreams; ++s) {
            const int sRow = starRow[s];
            const int nrows = starRow[s+1]-starRow[s];
        
            cuda_ret = hipBindTexture(NULL, xTex, v_d, n_global*sizeof(real));
            cuda_ret = hipBindTexture(NULL, valTex, vals_d, nnz_global*sizeof(real));
            
            spmv<<<grid[s], block[s], sharedMemorySize[s], stream[s] >>>((w_d+sRow), (rows_d+sRow), (cols_d), nrows);
            //spmv<<<grid, block, sharedMemorySize, stream[s] >>>((w_d+sRow),  v_d,  (vals_d), (rows_d+sRow), (cols_d), nrows);
            
            cuda_ret = hipUnbindTexture(xTex);
            cuda_ret = hipUnbindTexture(valTex);

        } // end for //
        
        
        for (int s=0; s<nStreams; ++s) {
            //hipStreamSynchronize(NULL);
            hipStreamSynchronize(stream[s]);
        } // end for //
        
        
    } // end for //
    
    gettimeofday(&tp,NULL);
    elapsed_time += (tp.tv_sec*1.0e6 + tp.tv_usec);
    printf ("Total time was %f seconds, GFLOPS: %f\n", elapsed_time*1.0e-6, 2.0*nnz_global*REP*1.0e-3/elapsed_time);

    cuda_ret = hipMemcpy(w, w_d, (n_global)*sizeof(real),hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device matrix y_d back to host");

// cuda stuff ends here
//////////////////////////////////////
   
    if (checkSol=='t') {
        real *sol=NULL;
        sol     = (real *) malloc((n_global)*sizeof(real)); 
        // reading input vector
        vectorReader(sol, &n_global, argv[3]);
        
        int row=0;
        const real tolerance=1.0e-08;
        real error;
        do {
            error =  fabs(sol[row] - w[row]) /fabs(sol[row]);
            if ( error > tolerance ) break;
            ++row;
        } while (row < n_global); // end do-while //
        
        if (row == n_global) {
            printf("Solution match in GPU\n");
        } else {    
            printf("For Matrix %s, solution does not match at element %d in GPU  %20.13e   -->  %20.13e  error -> %20.13e, tolerance: %20.13e \n", 
            argv[1], (row+1), sol[row], w[row], error , tolerance  );
        } // end if //
        free(sol);    
    } // end if //
    free(w);
    free(v);
    
    #include "parallelSpmvCleanData.h" 
    return 0;    
} // end main() //
