#include "hip/hip_runtime.h"
#include <stdio.h>
#include "real.h"
#include "parallelSpmv.h"

#ifdef DOUBLE
    extern texture<int2> xTex;
    //extern texture<int2> valTex;
#else
    extern texture<float> xTex;
    //extern texture<float> valTex;
#endif

#ifdef DOUBLE
    static __inline__ __device__ 
    double fetch_real(texture<int2> t, int i)
    {
	    int2 v = tex1Dfetch(t,i);
	    return __hiloint2double(v.y, v.x);
    } // end of fetch_real() //
#else
    static __inline__ __device__ 
    float fetch_real(texture<float> t, int i)
    {
	    return tex1Dfetch(t,i);
    } // end of fetch_double() //
#endif
__global__ 
void alg3   (real *__restrict__ const y, 
             const real *__restrict__ const val, 
             const int  *__restrict__ const col_idx, 
             const int  *__restrict__ const row_Ptr,
             const int  *__restrict__ const blockRows_d, 
             const int sizeBlockRows,
             const real alpha,
             const real beta
            )
{
    __shared__ real temp_s[SHARED_SIZE];

    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int starRow=blockRows_d[bid];
    int endRow=blockRows_d[bid+1];
    int firstCol=row_Ptr[starRow];
    int nnz = row_Ptr[endRow] - row_Ptr[starRow];
    
    
    for (int i=tid; i<nnz; i+=blockDim.x) {
        temp_s[i] = val[firstCol+i] * fetch_real( xTex, col_idx[firstCol+i]);
    } // end for //
    __syncthreads();

    if (tid < (endRow-starRow) ) {
        real sum = 0;
        int row_s = row_Ptr[starRow+tid]   - firstCol;
        int row_e = row_Ptr[starRow+tid+1] - firstCol;
        
        for (int i=row_s; i < row_e; ++i) {
            sum +=temp_s[i];
        } // end for //
        y[starRow+tid] =  beta*y[starRow+tid] +  alpha * sum;
    } // end if //

/*
    __shared__ int limit;
    __shared__ int row_Ptr_s[MAXTHREADS+1];
    __shared__ real temp_s[SHARED_SIZE];
    
    int tid = threadIdx.x;
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (row < nRows) {
        y[row] = beta*y[row];
        
        row_Ptr_s[tid] = row_Ptr[row];
        if (tid == 0) {
            limit = (nRows-row <  MAXTHREADS) ? nRows-row : MAXTHREADS; 
            row_Ptr_s[limit] = row_Ptr[row+limit];
        } // end if //
        
        __syncthreads();
        
        real sum=0.0;
        int toLoad=row_Ptr_s[limit]-row_Ptr_s[0] ;
        for (int i=row_Ptr_s[0]; i < row_Ptr_s[limit]; i+=SHARED_SIZE) {
        
            int index = tid + i;
            
            __syncthreads();

            for (int j=0;  j <= SHARED_SIZE/limit; ++j) {
                if (tid + j*limit <  (toLoad < SHARED_SIZE ? toLoad :SHARED_SIZE) ) {
                    temp_s[tid + j*limit] = temp[index];
                    index +=limit;
                } // end if //
            } // end for //
            
            __syncthreads();
            
            if (  row_Ptr_s[tid+1] > i  && row_Ptr_s[tid] <= (i+SHARED_SIZE-1) ) {
                int r_s = (row_Ptr_s[tid] - i > 0) ? row_Ptr_s[tid] - i : 0;
                int r_e = (row_Ptr_s[tid+1] - i < SHARED_SIZE) ? row_Ptr_s[tid+1] - i : SHARED_SIZE;
                for (int j=r_s; j < r_e; ++j) {
                    sum += temp_s[j];
                } // end for //
            } // end if //
            
            y[row] = alpha * sum;
            toLoad-=SHARED_SIZE;
        }  // end for //
    } // end if //    
*/    
} // end of alg3() //

/*

    if (tid==0)  printf("bid: %d starRow: %d, endRow: %d, nnz: %d\n", bid, starRow, endRow, nnz);


    if (tid==0)  printf("bid: %d starRow: %d, endRow: %d\n", bid, starRow, endRow);

        if (tid==0 ) {
            printf("tid: %d,blockIdx: %d, limit: %d [%d,%d]\n", tid, blockIdx.x, limit, row_Ptr_s[0], row_Ptr_s[limit]);
        }
*/        


/*
            if (tid==0 && blockIdx.x==0) {
                printf("index and toLoad : (%3d, %3d)---->  ", index, toLoad);
            }
*/


/*            
            if (tid==0 && blockIdx.x==0) {
                for (int j=0;  j < (toLoad < SHARED_SIZE ? toLoad :SHARED_SIZE); ++j) {
                    printf("%f, ", temp_s[j] );
                } // end for //            
                printf("\n");
            }
*/

        //if (blockIdx.x==gridDim.x-1) printf("%d, %d %d \n", tid,  row_Ptr_s[tid], row_Ptr_s[tid+1] );

